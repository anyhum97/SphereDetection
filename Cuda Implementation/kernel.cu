#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <vector>

#include <Eigen/Geometry>

#include "hip/hip_runtime.h"
#include ""

#include "Reflection.h"
#include "ply.h"

using namespace ply;

unsigned int vertex = 1024;

struct sphere
{
	ply::float3 center;

	float radius = 0.0f;
	float trust = 0.0f;
};

hipEvent_t start;
hipEvent_t stop;

void CudaMalloc()
{
    hipSetDevice(0);
}

void CudaFree()
{
    hipDeviceReset();
}

void Test()
{
    hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start, 0);

    ////////////////////////////////////////////////////////////////////////

    

    ////////////////////////////////////////////////////////////////////////

    hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	float time = 0;

	hipEventElapsedTime(&time, start, stop);

	hipEventDestroy(start);
	hipEventDestroy(stop);

	std::cout << time << "ms [OK]\n\n";
}

int main()
{
    CudaMalloc();

    ////////////////////////////////////////////////////////////////////////

	

    ////////////////////////////////////////////////////////////////////////

    CudaFree();
	return 0;
}