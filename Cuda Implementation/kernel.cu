#include <iostream>
#include <fstream>
#include <vector>

#include <Eigen/Geometry>

#include "hip/hip_runtime.h"
#include ""

#include "Reflection.cu"

hipEvent_t start;
hipEvent_t stop;

void CudaMalloc()
{
    hipSetDevice(0);
}

void CudaFree()
{
    hipDeviceReset();
}

void Test()
{
    hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start, 0);

    ////////////////////////////////////////////////////////////////////////

    

    ////////////////////////////////////////////////////////////////////////

    hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	float time = 0;

	hipEventElapsedTime(&time, start, stop);

	hipEventDestroy(start);
	hipEventDestroy(stop);

	std::cout << time << "ms [OK]\n\n";
}

void main()
{
    CudaMalloc();

    ////////////////////////////////////////////////////////////////////////

	

    ////////////////////////////////////////////////////////////////////////

    CudaFree();
}